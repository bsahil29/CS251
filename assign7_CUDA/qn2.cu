
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#include<math.h>
#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))
__device__ unsigned int myxor(unsigned long x,unsigned long y)
{
    unsigned int  result = 0;
    for (int  i = 63; i >= 0; i--)                     
    {
       // Find current bits in x and y
       bool b1 = x & (1 << i);
       bool b2 = y & (1 << i);
        
        // If both are 1 then 0 else xor is same as OR
        bool xoredBit = (b1 & b2) ? 0 : (b1 | b2);          
 
        // Update result
        result <<= 1;
        result |= xoredBit;
    }
    return result;
}
__global__ void calculate(unsigned long *mem, unsigned long num,int iter)
{
    int  i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i >= num)
           return;
    if(i<num/2){
		int temp=2*i;
		if(temp+iter<num)
			mem[temp]=myxor(mem[temp],mem[temp+iter]);
    }
}

int main(int argc, char **argv)
{
    struct timeval start, end, t_start, t_end;
    int i;
    unsigned long *ptr;
    unsigned long *gpu_mem;   
    unsigned long num,SEED;
    int  blocks=0;

    if(argc == 3){
         num = atoi(argv[1]);  
        SEED = atoi(argv[2]);
    }

    /* Allocate host (CPU) memory and initialize*/

    srand(SEED);
    ptr =(unsigned long*) malloc((num+1) * sizeof(unsigned long));
    for(i=0; i<num; ++i){
       ptr[i] = random();
    }
    ptr[i]=0;
    
    gettimeofday(&t_start, NULL);
    
    /* Allocate GPU memory and copy from CPU --> GPU*/

    hipMalloc(&gpu_mem, (num+1) * sizeof(unsigned long));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(gpu_mem, ptr, (num+1) * sizeof(unsigned long) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");
    
    gettimeofday(&start, NULL);
    
    blocks = (num) /1024;
    
    if((num) % 1024)
           ++blocks;
    for(i=0;i<log(num)/log(2);i++)
    {
    	calculate<<<blocks, 1024>>>(gpu_mem, num,(int)pow(2,i));
    }
    CUDA_ERROR_EXIT("kernel invocation");
    gettimeofday(&end, NULL);
    
    /* Copy back result*/
    hipMemcpy(ptr, gpu_mem, (num+1) * sizeof(unsigned long) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);
    
    printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(gpu_mem);
    /*Print the last element for sanity check*/ 
    printf("XOR sum=%lu\n",ptr[0]);
    free(ptr);
}